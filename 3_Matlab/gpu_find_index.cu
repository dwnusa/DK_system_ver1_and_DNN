#include "hip/hip_runtime.h"
#include "gpu_find_index.h"

__global__ void gpu_find_index_CUDA(bool* dst, double* origX, double* origY, double* thrX, double* thrY, int numRows_orig, int numRows_thr)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if(row >= numRows_orig)
        return;
    for(int col = 0; col < numRows_thr; col++)
    {
        if (origX[row] == thrX[col])
        {
            if (origY[row] == thrY[col])
            {
                dst[row] = true;
            }
        }
    }
}
void gpu_find_index(bool* dst, double* origX, double* origY, double* thrX, double* thrY, int numRows_orig, int numRows_thr)
{
    double* device_origX, *device_origY, *device_thrX, *device_thrY;
    bool* device_dst;
    
    hipMalloc(&device_origX, sizeof(double) * numRows_orig);
    hipMalloc(&device_origY, sizeof(double) * numRows_orig);
    hipMalloc(&device_thrX, sizeof(double) * numRows_thr);
    hipMalloc(&device_thrY, sizeof(double) * numRows_thr);
    hipMalloc(&device_dst, sizeof(bool) * numRows_orig);

    hipMemcpy(device_origX, origX, sizeof(double) * numRows_orig, hipMemcpyHostToDevice);
    hipMemcpy(device_origY, origY, sizeof(double) * numRows_orig, hipMemcpyHostToDevice);
    hipMemcpy(device_thrX, thrX, sizeof(double) * numRows_thr, hipMemcpyHostToDevice);
    hipMemcpy(device_thrY, thrY, sizeof(double) * numRows_thr, hipMemcpyHostToDevice);

    hipMemset(device_dst, false, sizeof(bool) * numRows_orig);

    dim3 blockSize(16*16);     
    dim3 gridSize((numRows_orig+(16*16-1))/blockSize.x);
    
    gpu_find_index_CUDA<<<gridSize,blockSize>>>(device_dst, device_origX, device_origY, device_thrX, device_thrY, numRows_orig, numRows_thr);

    hipMemcpy(dst, device_dst, sizeof(bool) * numRows_orig, hipMemcpyDeviceToHost);

    hipFree(device_origX);
    hipFree(device_origY);
    hipFree(device_thrX);
    hipFree(device_thrY);
    hipFree(device_dst);
}
